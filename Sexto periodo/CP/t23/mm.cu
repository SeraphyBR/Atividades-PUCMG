
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/*
    Tempo sequencial:
    real    1m6.953s
    user    1m6.791s
    sys     0m0.100s

    Tempo Paralelo CPU:
    real    0m17.541s
    user    1m9.719s
    sys     0m0.144s

    Tempo Paralelo GPU (distribute parallel for simd):
    real    0m5.409s
    user    0m3.535s
    sys     0m1.883s

    Nvprof: warps_launched          72          72          72          72
            warp_execution_efficiency                 Warp Execution Efficiency      13.10%      13.10%      13.10%

    Tempo Paralelo GPU (CUDA):
    real    0m0.883s
    user    0m0.028s
    sys     0m0.776s

    Nvprof: warps_launched        3200        3200        3200        3200
            warp_execution_efficiency                 Warp Execution Efficiency      93.77%      93.77%      93.77%

*/

__global__ void mm_cuda(double* a, double* b, double* c, int width) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    double sum = 0;
    if(i < width && j < width) {
        for (int k = 0; k < width; k++) {
            double x = a[i * width + k];
            double y = b[k * width + j];
            sum += x * y;
        }
        c[i * width + j] = sum;
    }
}

int main()
{
	int width = 2000;
    int size = width * width * sizeof(double);
	double *a = (double*) malloc (size);
	double *b = (double*) malloc (size);
	double *c = (double*) malloc (size);

	for(int i = 0; i < width; i++) {
		for(int j = 0; j < width; j++) {
			a[i*width+j] = i;
			b[i*width+j] = j;
			c[i*width+j] = 0;
		}
	}

    double *d_a, *d_b, *d_c;

    hipMalloc((void **) &d_a, size);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_b, size);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_c, size);

    int block_size = 32; // 32 * 32 = 1024;

    int num_blocks = (width - 1) / block_size + 1;
    dim3 dimGrid(num_blocks, num_blocks, 1);
    dim3 dimBlock(block_size, block_size, 1);

	mm_cuda<<<dimGrid, dimBlock>>>(d_a,d_b,d_c,width);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	//for(int i = 0; i < width; i++) {
	//    for(int j = 0; j < width; j++) {
	//        printf("\n c[%d][%d] = %f",i,j,c[i*width+j]);
	//    }
	//}

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
